#include <hip/hip_runtime.h>

__global__ void saxpy_kernel(int n, float a, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        y[i] = a * x[i] + y[i];
    }
}

void saxpy(int n, float a, float *x, float *y)
{
    float *d_x, *d_y; // Pointers for GPU memory

    // Allocate GPU memory for x and y
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, n * sizeof(float));

    // Copy data from CPU to GPU
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Define thread block size and compute grid dimensions
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch the SAXPY CUDA kernel
    saxpy_kernel<<<gridSize, blockSize>>>(n, a, d_x, d_y);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

    // Copy the result back from GPU to CPU
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_x);
    hipFree(d_y);
}
